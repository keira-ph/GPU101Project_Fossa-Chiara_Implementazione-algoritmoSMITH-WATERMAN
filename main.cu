
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define S_LEN 512
#define N 1000
#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }


double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

int max4(int n1, int n2, int n3, int n4)
{
    int tmp1, tmp2;
    tmp1 = n1 > n2 ? n1 : n2;
    tmp2 = n3 > n4 ? n3 : n4;
    tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
    return tmp1;
}
__device__  int max4GPU(int n1, int n2, int n3, int n4)
{
    int tmp1, tmp2;
    tmp1 = n1 > n2 ? n1 : n2;
    tmp2 = n3 > n4 ? n3 : n4;
    tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
    return tmp1;
}
void backtrace(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
    int n;

    for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
    {
        int dir = dir_mat[i][j];
        if (dir == 1 || dir == 2)
        {
            i--;
            j--;
        }
        else if (dir == 3)
            i--;
        else if (dir == 4)
            j--;

        simple_rev_cigar[n] = dir;
    }
    if(dir_mat[i][j] == 0)
        simple_rev_cigar[n] = 0;

}
__device__ void backtraceGPU(char *simple_rev_cigar, char *dir_mat, int maxind, int max_cigar_len) {

    int tid = threadIdx.x;
    int row = blockIdx.x;
    __shared__ int countonnext,prec;
  __shared__ int  x, righe, colonne;
    __shared__ int n,stop;
    __shared__ int next[1024];

    n=0;
    stop=0;
    __shared__ int partenza;
    partenza=((maxind+1)%(513*513))-1;




    __syncthreads();

   while ( n<2*512 && stop==0)
    {

        if(tid==0){
            righe=(partenza+1)/513+1;
            colonne=(partenza+1)%513;
            if(righe*colonne>=1024){
                if(righe>31 && colonne>31){
                    righe=32;
                    colonne=32;
                }else if(righe<=31){
                    colonne=1024/righe;
                }else if(colonne<=31){
                    righe=1024/colonne;
                }
            }
        }

        __syncthreads();

        if(tid<righe*colonne){
            x=partenza-(tid/colonne)*513- tid%colonne;
            int dir = dir_mat[row*513*513+x];
            if (dir == 1 || dir == 2){
               if(((partenza-x+1)/513)<righe-1 && ((partenza-x)%513)<colonne-1)
                   next[tid]=tid + colonne + 1;
                else
                next[tid]=1024;

            }
            else if (dir == 3){
                if(((partenza-x)/(513))<righe-1)
                    next[tid]=tid+colonne;
                else
                     next[tid]=1025;
            }
            else if (dir == 4){
                if((partenza-x)%513<colonne-1)
                next[tid]=tid+1;
                else
                next[tid]=1026;
            }

            else if (dir == 0)
                next[tid]=-1;


        }

        __syncthreads();
        /*if(tid==0 && next[0]==0 ){

            printf("%d-", next[0]);
            }*/




        if(tid==0){
            countonnext=0;
            prec=0;
            /*if(n==0 && row==0)
                printf("%d ",dir_mat[partenza]);*/
             while( countonnext >=0 && countonnext<(righe*colonne)){
                 simple_rev_cigar[n] = dir_mat[row*513*513+partenza-(countonnext/colonne)*513-(countonnext%colonne)];
                 prec=countonnext;
                 countonnext=next[countonnext];
                 n++;
             }
            if (countonnext!=-1 && countonnext!=1024 && countonnext!=1025 && countonnext!=1026){
                printf("ERR%d%d%dn", countonnext, prec,n);

            }

             if (countonnext==1024)
                 partenza=partenza-((prec)/colonne)*513-(prec)%colonne-514;
             else if (countonnext==1025)
                 partenza=partenza-((prec)/colonne)*513-(prec)%colonne-513;
             else if (countonnext==1026)
                 partenza=partenza-((prec)/colonne)*513-(prec)%colonne-1;
             else if (countonnext==-1){
                 stop=1;
                 simple_rev_cigar[n]=0;

             }
        }


        __syncthreads();

     }

    /*if(row==33 && tid==1023)
        printf("R%d-T%d ", row, tid);*/

}

__global__ void inplinGpu(char* gquery, char* greference, char* gsimple_rev_cigar, int *gres, int* sc_mat, char* dir_mat) {

    int tid = threadIdx.x;
    int row = blockIdx.x;

    /*if(row==999 && tid==1023)
        printf("R%d-T%d ", row, tid);*/
   __shared__ int max[1025];
    __shared__ int posizioni[1025];
    if(tid<1024)
      max[tid]=-2;
    if(tid==0){
       max[1024]=-2;

    }






    // initialize the scoring matrix and direction matrix to 0
    if(tid<1024){


        for (int j = 0; j < 257; j++)
        {
            sc_mat[(row*513*513)+(tid * 257 + j)] = 0;

            dir_mat[(row*513*513)+(tid * 257 + j)] = 0;
        }
        if(tid==0)
            sc_mat[(row*513*513)+((S_LEN+1)*(S_LEN+1)-1)] = 0;
        if(tid==1023)
            dir_mat[(row*513*513)+((S_LEN+1)*(S_LEN+1)-1)] = 0;
    }



    __syncthreads();


    // compute the alignment
    for (int n = 2; n<= 1024; n++) {
        if (tid>0 && tid < n && tid<=511 && (n-tid)<=511) {
            // compare the sequences characters
            int comparison = (gquery[(row)*S_LEN+(tid-1)] == greference[(row)*S_LEN+(n-tid-1)]) ? 1 : -1;
            // compute the cell knowing the comparison result
            int x=(row*513*513)+((tid)*513+(n-tid));
            int tmp = max4GPU(sc_mat[x-514] + comparison, sc_mat[x-513] -2, sc_mat[x-1] -2, 0);
            char dir;
            if (tmp == (sc_mat[x-514] + comparison))
                dir = comparison == 1 ? 1 : 2;
            else if (tmp == (sc_mat[x-513] -2))
                dir = 3;
            else if (tmp == (sc_mat[x-1] -2))
                dir = 4;
            else
                dir = 0;
            dir_mat[x] = dir;
            sc_mat[x] = tmp;

        }
        __syncthreads();
    }

   /* if(tid==0 && row==333){
        for(int i=0; i<100; i++)
            printf("GS%d-D%d ",sc_mat[333*513*513+333*513+i], dir_mat[333*513*513+333*513+i]);
    }*/

    int j;
    if(tid<1024){
        if(tid==0){
            max[1024]=sc_mat[(row*513*513)+(1024 * 257 )];
            posizioni[1024]=(row*513*513)+(1024 * 257 );

        }
        for ( j = 0; j < 257; j++)
        {
            if(sc_mat[(row*513*513)+(tid * 257 + j)]>max[tid]){
                max[tid]=sc_mat[(row*513*513)+(tid * 257 + j)];
                posizioni[tid]=(row*513*513)+(tid * 257 + j);
            }
        }


    }
    __syncthreads();


    if(tid==0){
        for(j=0; j<=1024; j++){
            if(max[j]>max[0]){
                max[0]=max[j];
                posizioni[0]=posizioni[j];
            }
        }



    }/*
    if(tid==0){
        for(int t=0; t<513*513;  t++){
            if(gres[row]<sc_mat[(row*513*513)+t])
                gres[row]=sc_mat[(row*513*513)+t];
        }
    }*/
    gres[row] = max[0];


    __syncthreads();


    backtraceGPU(&gsimple_rev_cigar[row*512*2], dir_mat, posizioni[0], 512 * 2);


}

int main(int argc, char *argv[])
{
    srand(time(NULL));

    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};

    int ins = -2, del = -2, match = 1, mismatch = -1; // penalties

    char **query = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        query[i] = (char *)malloc(S_LEN * sizeof(char));

    char **reference = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        reference[i] = (char *)malloc(S_LEN * sizeof(char));

    int **sc_mat = (int **)malloc((S_LEN + 1) * sizeof(int *));
    for (int i = 0; i < (S_LEN + 1); i++)
        sc_mat[i] = (int *)malloc((S_LEN + 1) * sizeof(int));
    char **dir_mat = (char **)malloc((S_LEN + 1) * sizeof(char *));
    for (int i = 0; i < (S_LEN + 1); i++)
        dir_mat[i] = (char *)malloc((S_LEN + 1) * sizeof(char));

    int *res = (int *)malloc(N * sizeof(int));
    char **simple_rev_cigar = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        simple_rev_cigar[i] = (char *)malloc(S_LEN * 2 * sizeof(char));

    char *Q = (char *)malloc(N*S_LEN * sizeof(char));
    char *R = (char *)malloc(N*S_LEN * sizeof(char));
    int *RES = (int *)malloc(N * sizeof(int));
    char *SRC = (char *)malloc(N*2*S_LEN * sizeof(char));


    // randomly generate sequences
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < S_LEN; j++)
        {
            query[i][j] = alphabet[rand() % 5];

            Q[i*S_LEN+j]=query[i][j];
            reference[i][j] = alphabet[rand() % 5];
            R[i*S_LEN+j]=reference[i][j];
        }
    }

    double start_cpu = get_time();

    for (int n = 0; n < N; n++)
    {
        int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
       int maxi, maxj;
        // initialize the scoring matrix and direction matrix to 0
        for (int i = 0; i < S_LEN + 1; i++)
        {
            for (int j = 0; j < S_LEN + 1; j++)
            {
                sc_mat[i][j] = 0;
                dir_mat[i][j] = 0;
            }
        }
        // compute the alignment
        for (int i = 1; i < S_LEN; i++)
        {
            for (int j = 1; j < S_LEN; j++)
            {
                // compare the sequences characters
                int comparison = (query[n][i - 1] == reference[n][j - 1]) ? match : mismatch;
                // compute the cell knowing the comparison result
                int tmp = max4(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);
                char dir;

                if (tmp == (sc_mat[i - 1][j - 1] + comparison))
                    dir = comparison == match ? 1 : 2;
                else if (tmp == (sc_mat[i - 1][j] + del))
                    dir = 3;
                else if (tmp == (sc_mat[i][j - 1] + ins))
                    dir = 4;
                else
                    dir = 0;

                dir_mat[i][j] = dir;
                sc_mat[i][j] = tmp;

                if (tmp > max)
                {
                    max = tmp;
                    maxi = i;
                    maxj = j;
                }
            }
        }

        res[n] = sc_mat[maxi][maxj];


        backtrace(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);


    }
    /*printf("%d",simple_rev_cigar[0][0]);*/
    double end_cpu = get_time();
    printf("SW Time CPU: %.10lf\n", end_cpu - start_cpu);
    char *gquery, *greference,*gdir ,*gsimple_rev_cigar;
    int *gsc, *gres;

    CHECK(hipMalloc(&gquery, N*S_LEN * sizeof(char)));
    CHECK(hipMalloc(&greference, N*S_LEN * sizeof(char)));

    CHECK(hipMalloc(&gsimple_rev_cigar, 2*N*S_LEN * sizeof(char)));
    CHECK(hipMalloc(&gres, N * sizeof(int)));

    CHECK(hipMalloc(&gsc, N*(S_LEN + 1) * (S_LEN + 1) * sizeof(int)));
    CHECK(hipMalloc(&gdir, N*(S_LEN + 1) * (S_LEN + 1) * sizeof(char)));



    CHECK(hipMemcpy(gquery, Q, N*S_LEN * sizeof(char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(greference, R, N*S_LEN * sizeof(char), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(1000, 1, 1);
    dim3 threadsPerBlock(1024, 1, 1);

    double start_gpu = get_time();

    inplinGpu<<<blocksPerGrid, threadsPerBlock>>>(gquery,greference,gsimple_rev_cigar,gres,gsc, gdir);


    double end_gpu = get_time();


    printf("SW Time GPU: %.10lf\n", end_gpu - start_gpu);

    CHECK(hipMemcpy(RES, gres, N * sizeof(int), hipMemcpyDeviceToHost));

    CHECK(hipMemcpy(SRC,gsimple_rev_cigar, N*2*S_LEN* sizeof(char), hipMemcpyDeviceToHost));
    int t;
    for( t=0;t<N && res[t]==RES[t];t++);
        if(t==N)
        printf("VERIFICA!\n");

    for( int l=0;l<N;l++){
        for( t=0;SRC[512*2*l+t]!=0;t++){
            if(SRC[512*2*l+t]!=simple_rev_cigar[l][t])
                printf("LLL%d,%d,%d,%d\n",SRC[512*2*l+t],simple_rev_cigar[l][t],l,t);

        }


    }



    return 0;
}